#include "hip/hip_runtime.h"
#include "upsampling.h"
#include <iostream>
#include <cassert>

// gpu operation for nearest neighbor upsampling
template <typename T>
__global__ void gpuResizeNearestNeighbor( T* input, int nChannels, int iHeight, int iWidth, T* output)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int z = blockIdx.z * blockDim.z + threadIdx.z;

    const size_t oWidth = 2 * iWidth;
    const size_t oHeight = 2 * iHeight;

    if( x >= nChannels || y >= oHeight || z >= oWidth )
        return;

    const int dy = ((float)y * 0.5);
    const int dz = ((float)z * 0.5);

    const T px = input[x * iWidth * iHeight + dy * iWidth + dz];

    output[x * oWidth * oHeight + y * oWidth + z] = px;
}


// nearest neighbor upsampling
template <typename T>
hipError_t cudaResizeNearestNeighbor( T* input, size_t nChannels, size_t inputWidth, size_t inputHeight,
                        T* output, hipStream_t stream )
{
    std::cout << "cudaResizeNearestNeighbor" << std::endl;
    if( !input || !output )
    {
        std::cout << "No input or no output" << std::endl;
        return hipErrorInvalidDevicePointer;
    }

    if( inputWidth == 0 || inputHeight == 0 )
    {
        std::cout << "Width or height is 0" << std::endl;
        return hipErrorInvalidValue;
    }

    // launch kernel
    const dim3 blockDim(4, 8, 8);
    const size_t outputWidth = 2 * inputWidth;
    const size_t outputHeight = 2 * inputHeight;
    const dim3 gridDim(iDivUp(nChannels, blockDim.x), iDivUp(outputHeight, blockDim.y), iDivUp(outputWidth, blockDim.z));

    gpuResizeNearestNeighbor<T><<<gridDim, blockDim, 0, stream>>>(input, nChannels, inputHeight, inputWidth, output);

    return CUDA(hipGetLastError());
}

//gpu operation for bilinear upsampling
template <typename T>
// TODO
__global__ void gpuResizeBilinear( float2 scale, T* input, int iWidth, T* output, int oWidth, int oHeight ) { }

// bilinear upsampling
hipError_t cudaResizeBilinear( float* input, size_t inputWidth, size_t inputHeight,
                        float* output, size_t outputWidth, size_t outputHeight )
{
    // TODO
    return CUDA(hipGetLastError());
}

template hipError_t cudaResizeNearestNeighbor<float>(float*, size_t, size_t, size_t, float*, hipStream_t);
template hipError_t cudaResizeNearestNeighbor<__half>(__half*, size_t, size_t, size_t, __half*, hipStream_t);